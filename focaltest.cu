#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>

#include "CuLayer.h"

#include "CuEnvControl.h"
#include "LocalOperator.h"
#include "FocalOperator.h"
#include "NeighborhoodSlope.h"
#include <time.h>

#include "cputest.h"

using namespace std;
using namespace CuPRL;



int main(int argc, char* argv[])
{

	/*
	int testdata[36] = { 1, 1, 1, 1, 1, 1,
		1, 3, 3, 2, 1, 10,
		1, 1, 3, 2, 2, 2,
		1, 2, 2, 2, 2, 2,
		1, 1, 1, 2, 2, 2,
		1, 1, 1, 1, 1, 2 };


	CuLayer<int>testlayer1(testdata, 6, 6);
	testlayer1.setCellHeight(1);
	testlayer1.setCellWidth(1);
	testlayer1.setNoDataValue(10);

	printLayer(testlayer1);
	*/
	//CuEnvControl::setBlockDim(16, 16);

	CuLayer<int>testlayer1;
	testlayer1.Read("C:\\Users\\HP\\Desktop\\WH-DEM\\focalTest\\focalTest1.tif");

	//testlayer1.Read("C:\\Users\\HP\\Desktop\\WH-DEM\\focalTest\\TestRasterWH.tif");
	
	/*int max = 0;
	for (int i = 0; i < testlayer1.getHeight(); i++)
	{
		for (int j = 0; j < testlayer1.getWidth(); j++)
		{
			if (testlayer1[i*testlayer1.getWidth() + j] > max&&testlayer1[i*testlayer1[i*testlayer1.getWidth()+j]]!=10000)
			{
				max = testlayer1[i*testlayer1.getWidth() + j];
			}
		}
	}
	cout << "max=" << max << endl;
	*/
	//cout << testlayer1[0] << endl;



	clock_t t1, t2;

	t1 = clock();

	NeighborhoodSlope neiSlope;

	//CuLayer<double>testgpulayer = cuFocalOperatorFn<int, double, int, SlopeCal>(testlayer1, &neiSlope);


	NeighborhoodRect<int>neiRect(3, 3);

	//CuLayer<int>testSumLayer = testlayer1;
	CuLayer<float>testSumLayer = focalStatisticsMean<int, float, int>(testlayer1, &neiRect, NOUSE, IGNORE);

	t2 = clock();

	cout << t2 - t1 << endl;

	t1 = clock();
	
	
	//CuLayer<double>testcpulayer = CPUSlopeCal(testlayer1, &neiSlope);

	t2 = clock();

	cout << t2 - t1 << endl;
	/*
	if (compareLayer(testcpulayer, testgpulayer) == false)
	{
		cout << "result error" << endl;
	}
	else
	{
		cout << "result right" << endl;
	}
	*/
	/*
	int t = 58 + 3251 * testlayer1.getWidth();

	for (int i = -1; i < 1; i++)
	{
		for (int j = -1; j <= 1; j++)
		{
			cout << testlayer1[t + i*testlayer1.getWidth() + j] << " ";
		}
		cout << endl;
	}
	*/
	testSumLayer.Write("C:\\Users\\HP\\Desktop\\WH-DEM\\focalTest\\TestResult.tif");

	system("pause");
	return 0;
}